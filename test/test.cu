#include "hip/hip_runtime.h"
#include "test.hpp"



int max_cells;

int64_t serialTimeMean = 0, ompTimeMean = 0, cudaTimeMean = 0;

int main() {
    std::ofstream file;

    file.open("results.csv");
    file << "Matrix size,Serial,OMP,Speedup,Efficiency\n";

    const int iterations = 15;
    const int maxSize = 800;
    
    for (int matrixSize = 100; matrixSize <= maxSize; matrixSize+= 50)
    {
        // SETUP ----------------------------------------------------------------------------------------------
        std::cout<<"------------------------------------------------------------------"<<std::endl;
        double dt = 0.1;
        double diff = 0.1;

        double speedup, efficiency;

        std::srand(unsigned(std::time(nullptr)));

        std::vector<float> value(matrixSize * matrixSize);
        std::generate(value.begin(), value.end(), randFloat);

        std::vector<float> oldValue(matrixSize * matrixSize);
        std::generate(oldValue.begin(), oldValue.end(), randFloat);
        std::fill(oldValue.begin(), oldValue.end(), 0);


        std::vector<float> valueOmp(matrixSize * matrixSize);
        std::copy(value.begin(), value.end(), valueOmp.begin());

        std::vector<float> oldValueOmp(matrixSize * matrixSize);
        std::copy(oldValue.begin(), oldValue.end(), oldValueOmp.begin());

        // SERIAL ----------------------------------------------------------------------------------------------
        std::cout << BOLD BLUE "Matrix size: " << RESET << matrixSize << std::endl;
        for (int i = 0; i < iterations; i++) {
            auto serialBegin = std::chrono::high_resolution_clock::now();
            diffuse(matrixSize, Axis::ZERO, value, oldValue, diff, dt);
            auto serialEnd = std::chrono::high_resolution_clock::now();
            auto serialTime = std::chrono::duration_cast<std::chrono::milliseconds>(serialEnd - serialBegin).count();

            serialTimeMean += serialTime;


        }

        serialTimeMean /= iterations;
        std::cout << BOLD YELLOW "Diffuse: " << serialTimeMean << RESET " millis "<<std::endl<<std::endl;

        file << matrixSize << "," << serialTimeMean << ",";


        // OMP ----------------------------------------------------------------------------------------------
        int max_threads = omp_get_max_threads();   
        omp_set_num_threads(max_threads);
        // Calculate how many cells as maximum per thread

        const int max_rows = (int)(ceil((matrixSize-2) / max_threads) + 2);
        max_cells = max_rows * (matrixSize-2);        

        int64_t ompTimeMean = 0;

        for (int i = 0; i < iterations; i++) {
            auto ompBegin = std::chrono::high_resolution_clock::now();
            omp_diffuse(matrixSize, Axis::ZERO, valueOmp, oldValueOmp, diff, dt);
            auto ompEnd = std::chrono::high_resolution_clock::now();
            auto ompTime = std::chrono::duration_cast<std::chrono::milliseconds>(ompEnd - ompBegin).count();

            ompTimeMean += ompTime;
        }
        ompTimeMean /= iterations;
        
        std::cout << BOLD RED "OMP Diffuse: " << ompTimeMean << RESET " millis" << std::endl;

        speedup = (double) serialTimeMean / (double) ompTimeMean;
        efficiency = speedup / max_threads;
        std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " ";
        std::cout << BOLD GREEN "Efficiency: " << RESET << efficiency << std::endl << std::endl;

        speedup = (double) serialTimeMean / (double) ompTimeMean;
        efficiency = speedup / max_threads;
        file << std::fixed << std::setprecision(2) << ompTimeMean << "," << std::setprecision(2) << speedup << "," << std::setprecision(2) << efficiency << "\n";
        
        printf("Speedup: %f\n", speedup);
        // // CUDA ----------------------------------------------------------------------------------------------
        // for (int i = 0; i < iterations; i++) {
        //     cuda_diffuse(matrixSize, Axis::ZERO, valueOmp, oldValueOmp, diff, dt);
        // }

        // cudaTimeMean /= iterations;
        
        // std::cout << BOLD GREEN "CUDA Diffuse: " << cudaTimeMean << RESET " millis" << std::endl;

        // speedup = (double) serialTimeMean / (double) cudaTimeMean;
        // std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " "<<std::endl<<std::endl;




        // std::cout << std::endl << std::endl;
    }



    return 0;
}

float randFloat() {
    return ((double) rand() / (RAND_MAX));
}

bool float_equals(float a, float b, float epsilon)
{
    return std::abs(a - b) < epsilon;
}

void diffuse(int N, Axis mode, std::vector<float> &value, std::vector<float> &oldValue, float diffusion, float dt) {
    float diffusionRate = dt * diffusion * N * N;

    lin_solve(N, mode, value, oldValue, diffusionRate);
}

void omp_diffuse(int N, Axis mode, std::vector<float> &value, std::vector<float> &oldValue, float diffusion, float dt) {
    float diffusionRate = dt * diffusion * N * N;

    omp_lin_solve(N, mode, value, oldValue, diffusionRate);
}

void cuda_diffuse(int N, Axis mode, std::vector<float> &value, std::vector<float> &oldValue, float diffusion, float dt) {
    float diffusionRate = dt * diffusion * N * N;
    
    dim3 BlockSize(16, 16, 1);
    dim3 GridSize((N+15)/16, (N+15)/16, 1);

    hipEvent_t cudaStart, cudaStop;	
    float milliseconds = 0;

    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaStop);

    float* d_value;
    float* d_oldValue;

    hipMalloc(&d_value, N * N * sizeof(float));
    hipMalloc(&d_oldValue, N * N * sizeof(float));

    hipMemcpy(d_value, &value[0], N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_oldValue, &oldValue[0], N * N * sizeof(float), hipMemcpyHostToDevice);

    std::cout<<"Hello from host"<<std::endl;

    hipEventRecord(cudaStart);    
    kernel_lin_solve<<<GridSize, BlockSize>>>(N, mode, &value[0], &oldValue[0], diffusionRate);
    hipEventRecord(cudaStop);

    hipMemcpy(&value[0], d_value, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(cudaStop);
    hipEventElapsedTime(&milliseconds, cudaStart, cudaStop);

    printf("Time for the kernel: %f ms\n", milliseconds);
    cudaTimeMean += milliseconds;


    hipFree(d_value);
    hipFree(d_oldValue);

    hipEventDestroy(cudaStart);
    hipEventDestroy(cudaStop);
}



void lin_solve(int N, Axis mode, std::vector<float> &nextValue, std::vector<float> &value, float diffusionRate) {
    float c = 1 + 4 * diffusionRate;
    float cRecip = 1.0 / c;
    for (int k = 0; k < ITERATIONS; k++) {
        for (int j = 1; j < N - 1; j++) {
            for (int i = 1; i < N - 1; i++) {
                nextValue[IX(i, j)] = (value[IX(i, j)]
                                   + diffusionRate * (
                        nextValue[IX(i + 1, j)]
                        + nextValue[IX(i - 1, j)]
                        + nextValue[IX(i, j + 1)]
                        + nextValue[IX(i, j - 1)]
                )) * cRecip;
            }
        }
        set_bnd(N, mode, nextValue);
    }
}

void omp_lin_solve(int N, Axis mode, std::vector<float> &nextValue, std::vector<float> &value, float diffusionRate) {
    float c = 1 + 4 * diffusionRate;
    float cRecip = 1.0 / c;
    for (int k = 0; k < ITERATIONS; k++)
    {
        #pragma omp parallel for default(shared) schedule(guided) collapse(2)
            for (int j = 1; j < N - 1; j++)
            {
                for (int i = 1; i < N - 1; i++)
                {
                    nextValue[IX(i, j)] = (value[IX(i, j)]
                                    + diffusionRate * (
                            nextValue[IX(i + 1, j)]
                            + nextValue[IX(i - 1, j)]
                            + nextValue[IX(i, j + 1)]
                            + nextValue[IX(i, j - 1)]
                    )) * cRecip;
                }
            }
        set_bnd(N, mode, nextValue);
    }
}

__global__ void kernel_lin_solve(int N, Axis mode, float* nextValue, float* value, float diffusionRate) {


    float c = 1 + 4 * diffusionRate;
    float cRecip = 1.0 / c;
    
    int col = threadIdx.x+blockIdx.x*blockDim.x;
	int row = threadIdx.y+blockIdx.y*blockDim.y;

	if(col == 0 || col >= N - 1 || row == 0 || row >= N - 1) return;

    printf("Hello from col: %d row: %d\n", col, row);

    for (int k = 0; k < ITERATIONS; k++) {
        nextValue[IX(row, col)] = (value[IX(row,col)]
                            + diffusionRate * (
                nextValue[IX(row + 1, col)]
                + nextValue[IX(row - 1, col)]
                + nextValue[IX(row, col + 1)]
                + nextValue[IX(row, col - 1)]
        )) * cRecip;

        // __syncthreads();
        // if (col == 1 && row == 1)
        //     kernel_set_bnd(N, mode, nextValue);
        // __syncthreads();
    }
}


void set_bnd(int N, Axis mode, std::vector<float> &attr) {
    for (int i = 1; i < N - 1; i++) {
        attr[IX(i, 0    )] = mode == Axis::Y ? -attr[IX(i, 1)] : attr[IX(i, 1)];
        attr[IX(i, N - 1)] = mode == Axis::Y ? -attr[IX(i, N - 2)] : attr[IX(i, N - 2)];
    }
    for (int j = 1; j < N - 1; j++) {
        attr[IX(0, j    )] = mode == Axis::X ? -attr[IX(1, j)] : attr[IX(1, j)];
        attr[IX(N - 1, j)] = mode == Axis::X ? -attr[IX(N - 2, j)] : attr[IX(N - 2, j)];
    }


    attr[IX(0    , 0    )] = 0.5f * (attr[IX(1, 0)] + attr[IX(0, 1)]);
    attr[IX(0    , N - 1)] = 0.5f * (attr[IX(1, N - 1)] + attr[IX(0, N - 2)]);

    attr[IX(N - 1, 0    )] = 0.5f * (attr[IX(N - 2, 0)] + attr[IX(N - 1, 1)]);
    attr[IX(N - 1, N - 1)] = 0.5f * (attr[IX(N - 2, N - 1)] + attr[IX(N - 1, N - 2)]);
}

__device__ void kernel_set_bnd(int N, Axis mode, float *attr) {
    for (int i = 1; i < N - 1; i++) {
        attr[IX(i, 0    )] = mode == Axis::Y ? -attr[IX(i, 1)] : attr[IX(i, 1)];
        attr[IX(i, N - 1)] = mode == Axis::Y ? -attr[IX(i, N - 2)] : attr[IX(i, N - 2)];
    }
    for (int j = 1; j < N - 1; j++) {
        attr[IX(0, j    )] = mode == Axis::X ? -attr[IX(1, j)] : attr[IX(1, j)];
        attr[IX(N - 1, j)] = mode == Axis::X ? -attr[IX(N - 2, j)] : attr[IX(N - 2, j)];
    }


    attr[IX(0    , 0    )] = 0.5f * (attr[IX(1, 0)] + attr[IX(0, 1)]);
    attr[IX(0    , N - 1)] = 0.5f * (attr[IX(1, N - 1)] + attr[IX(0, N - 2)]);

    attr[IX(N - 1, 0    )] = 0.5f * (attr[IX(N - 2, 0)] + attr[IX(N - 1, 1)]);
    attr[IX(N - 1, N - 1)] = 0.5f * (attr[IX(N - 2, N - 1)] + attr[IX(N - 1, N - 2)]);
}