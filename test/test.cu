#include "hip/hip_runtime.h"
#include "test.hpp"



int max_cells;

int64_t serialTimeMean = 0, ompTimeMean = 0, cudaTimeMean = 0;

int main() {
    std::ofstream file;

    const int iterations = 15;
    const int maxSize = 1200;

    testDiffuse(maxSize, iterations);



    return 0;
}

void testDiffuse(int maxSize, int iterations) {    
    std::ofstream file;
    file.open("diffuse_results.csv");
    file << "Matrix size,Serial,OMP,Speedup,Efficiency,Thread number\n";

    for (int matrixSize = 75; matrixSize <= maxSize; matrixSize+= 75)
    {
        // SETUP ----------------------------------------------------------------------------------------------
        std::cout<<"------------------------------------------------------------------"<<std::endl;
        double dt = 0.2;
        double diff = 0.0;
        double visc =  0.0000001f;

        double speedup, efficiency;

        std::srand(unsigned(std::time(nullptr)));

        std::vector<double> value(matrixSize * matrixSize);
        std::generate(value.begin(), value.end(), randdouble);

        std::vector<double> oldValue(matrixSize * matrixSize);
        std::fill(oldValue.begin(), oldValue.end(), 0);


        std::vector<double> valueOmp(matrixSize * matrixSize);
        std::copy(value.begin(), value.end(), valueOmp.begin());

        std::vector<double> oldValueOmp(matrixSize * matrixSize);
        std::copy(oldValue.begin(), oldValue.end(), oldValueOmp.begin());

        // SERIAL ----------------------------------------------------------------------------------------------
        std::cout << BOLD BLUE "Matrix size: " << RESET << matrixSize << std::endl;
        for (int i = 0; i < iterations; i++) {
            auto serialBegin = std::chrono::high_resolution_clock::now();
            diffuse(matrixSize, Axis::ZERO, value, oldValue, diff, dt);
            auto serialEnd = std::chrono::high_resolution_clock::now();
            auto serialTime = std::chrono::duration_cast<std::chrono::microseconds>(serialEnd - serialBegin).count();

            serialTimeMean += serialTime;


        }

        serialTimeMean /= iterations;
        std::cout << BOLD YELLOW "Diffuse: " << serialTimeMean << RESET " millis "<<std::endl<<std::endl;

        file << matrixSize << "," << serialTimeMean << ",";


        // OMP ----------------------------------------------------------------------------------------------
        int num_threads = omp_get_max_threads();
        omp_set_num_threads(num_threads);
        int threadMean = 0;
        int realThreadNumber = 0;

        // Calculate how many cells as maximum per thread
        const int max_rows = (int)(ceil((matrixSize-2) / num_threads) + 2);
        max_cells = max_rows * (matrixSize-2);        

        int64_t ompTimeMean = 0;

        for (int i = 0; i < iterations; i++) {    
            auto ompBegin = std::chrono::high_resolution_clock::now();
            omp_diffuse(matrixSize, Axis::ZERO, valueOmp, oldValueOmp, diff, dt, &realThreadNumber);
            auto ompEnd = std::chrono::high_resolution_clock::now();
            auto ompTime = std::chrono::duration_cast<std::chrono::microseconds>(ompEnd - ompBegin).count();
            threadMean += realThreadNumber;
            ompTimeMean += ompTime;
        }
        threadMean /= iterations;
        ompTimeMean /= iterations;
        
        std::cout << BOLD RED "OMP Diffuse: " << ompTimeMean << RESET " micros" << std::endl;

        speedup = (double) serialTimeMean / (double) ompTimeMean;
        efficiency = speedup / num_threads;
        std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " ";
        std::cout << BOLD GREEN "Efficiency: " << RESET << efficiency << std::endl << std::endl;

        file << std::fixed << std::setprecision(2) << ompTimeMean << "," << std::setprecision(2) << speedup << "," << std::setprecision(2) << efficiency << "," << threadMean << "\n";
        
        printf("Speedup: %f\n", speedup);
        // CUDA ----------------------------------------------------------------------------------------------
        /*for (int i = 0; i < iterations; i++) {
            cuda_diffuse(matrixSize, Axis::ZERO, valueOmp, oldValueOmp, diff, dt);
        }

        cudaTimeMean /= iterations;
        
        std::cout << BOLD GREEN "CUDA Diffuse: " << cudaTimeMean << RESET " millis" << std::endl;

        speedup = (double) serialTimeMean / (double) cudaTimeMean;
        std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " "<<std::endl<<std::endl;




        std::cout << std::endl << std::endl;*/
    }
}

void testAdvect(int maxSize, int iterations) {    
    std::ofstream file;
    file.open("advect_results.csv");
    file << "Matrix size,Serial,OMP,Speedup,Efficiency,Thread number\n";

    for (int matrixSize = 75; matrixSize <= maxSize; matrixSize+= 75)
    {
        // SETUP ----------------------------------------------------------------------------------------------
        std::cout<<"------------------------------------------------------------------"<<std::endl;
        double dt = 0.2;
        double diff = 0.0;
        double visc =  0.0000001f;

        double speedup, efficiency;

        std::srand(unsigned(std::time(nullptr)));

        
        // ------- Serial --------
        // Value
        std::vector<double> value(matrixSize * matrixSize);
        std::generate(value.begin(), value.end(), randdouble);

        // OldValue
        std::vector<double> oldValue(matrixSize * matrixSize);
        std::fill(oldValue.begin(), oldValue.end(), 0);
        

        // vX
        std::vector<double> vX(matrixSize * matrixSize);
        std::generate(vX.begin(), vX.end(), randdouble);

        // vX0
        std::vector<double> vX0(matrixSize * matrixSize);
        std::fill(vX0.begin(), vX0.end(), 0);
        
        // vY
        std::vector<double> vY(matrixSize * matrixSize);
        std::generate(vY.begin(), vY.end(), randdouble);

        // vY0
        std::vector<double> vY0(matrixSize * matrixSize);
        std::fill(vY0.begin(), vY0.end(), 0);
        

        // ------- OMP -------- 
        // Value
        std::vector<double> valueOmp(matrixSize * matrixSize);
        std::copy(value.begin(), value.end(), valueOmp.begin());

        // OldValue
        std::vector<double> oldValueOmp(matrixSize * matrixSize);
        std::copy(oldValue.begin(), oldValue.end(), oldValueOmp.begin());

        
        // vX
        std::vector<double> vXOmp(matrixSize * matrixSize);
        std::copy(vX.begin(), vX.end(), vXOmp.begin());

        // vX0
        std::vector<double> vX0Omp(matrixSize * matrixSize);
        std::fill(vX0Omp.begin(), vX0Omp.end(), 0);
        
        // vY
        std::vector<double> vYOmp(matrixSize * matrixSize);
        std::copy(vY.begin(), vY.end(), vYOmp.begin());

        // vY0
        std::vector<double> vY0Omp(matrixSize * matrixSize);
        std::fill(vY0Omp.begin(), vY0Omp.end(), 0);

        // SERIAL ----------------------------------------------------------------------------------------------
        std::cout << BOLD BLUE "Matrix size: " << RESET << matrixSize << std::endl;
        for (int i = 0; i < iterations; i++) {
            auto serialBegin = std::chrono::high_resolution_clock::now();

            advect(matrixSize, Axis::ZERO, vX, vX0, vX0, vY0, dt);

            auto serialEnd = std::chrono::high_resolution_clock::now();
            auto serialTime = std::chrono::duration_cast<std::chrono::microseconds>(serialEnd - serialBegin).count();

            serialTimeMean += serialTime;
        }

        serialTimeMean /= iterations;
        std::cout << BOLD YELLOW "Advect: " << serialTimeMean << RESET " micros "<<std::endl<<std::endl;

        file << matrixSize << "," << serialTimeMean << ",";


        // OMP ----------------------------------------------------------------------------------------------
        int num_threads = omp_get_max_threads();
        int realThreadNum = 0;
        omp_set_num_threads(num_threads);

        // Calculate how many cells as maximum per thread
        const int max_rows = (int)(ceil((matrixSize-2) / num_threads) + 2);
        max_cells = max_rows * (matrixSize-2);        

        int64_t ompTimeMean = 0;
        int threadNumMean = 0;
        for (int i = 0; i < iterations; i++) {
            auto ompBegin = std::chrono::high_resolution_clock::now();

            omp_advect(matrixSize, Axis::ZERO, vXOmp, vX0Omp, vX0Omp, vY0Omp, dt, &realThreadNum);

            auto ompEnd = std::chrono::high_resolution_clock::now();
            auto ompTime = std::chrono::duration_cast<std::chrono::microseconds>(ompEnd - ompBegin).count();
            
            threadNumMean += realThreadNum;
            ompTimeMean += ompTime;
        }
        threadNumMean /= iterations;
        ompTimeMean /= iterations;
        
        std::cout << BOLD RED "OMP Advect: " << ompTimeMean << RESET " micros" << std::endl;

        std::cout << BOLD PURPLE "Average team's threads number: " << threadNumMean << RESET << std::endl;

        speedup = (double) serialTimeMean / (double) ompTimeMean;
        efficiency = speedup / threadNumMean;
        std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " ";
        std::cout << BOLD GREEN "Efficiency: " << RESET << efficiency << std::endl << std::endl;

        file << std::fixed << std::setprecision(2) << ompTimeMean << "," << std::setprecision(2) << speedup << "," << std::setprecision(2) << efficiency << "," << threadNumMean << "\n";
        
        printf("Speedup: %f\n", speedup);
        // // CUDA ----------------------------------------------------------------------------------------------
        // for (int i = 0; i < iterations; i++) {
        //     cuda_diffuse(matrixSize, Axis::ZERO, valueOmp, oldValueOmp, diff, dt);
        // }

        // cudaTimeMean /= iterations;
        
        // std::cout << BOLD GREEN "CUDA Advect: " << cudaTimeMean << RESET " millis" << std::endl;

        // speedup = (double) serialTimeMean / (double) cudaTimeMean;
        // std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " "<<std::endl<<std::endl;




        // std::cout << std::endl << std::endl;
    }
}

void testProject(int maxSize, int iterations) {
    std::ofstream file;
    file.open("project_results.csv");
    file << "Matrix size,Serial,OMP,Speedup,Efficiency,Thread number\n";

    for (int matrixSize = 75; matrixSize <= maxSize; matrixSize+= 75)
    {
        // SETUP ----------------------------------------------------------------------------------------------
        std::cout<<"------------------------------------------------------------------"<<std::endl;
        double dt = 0.2;
        double diff = 0.0;
        double visc =  0.0000001f;

        double speedup, efficiency;

        std::srand(unsigned(std::time(nullptr)));

        std::vector<double> value(matrixSize * matrixSize);
        std::generate(value.begin(), value.end(), randdouble);

        std::vector<double> oldValue(matrixSize * matrixSize);
        std::fill(oldValue.begin(), oldValue.end(), 0);


        std::vector<double> valueOmp(matrixSize * matrixSize);
        std::copy(value.begin(), value.end(), valueOmp.begin());

        std::vector<double> oldValueOmp(matrixSize * matrixSize);
        std::copy(oldValue.begin(), oldValue.end(), oldValueOmp.begin());

        // SERIAL ----------------------------------------------------------------------------------------------
        std::cout << BOLD BLUE "Matrix size: " << RESET << matrixSize << std::endl;
        for (int i = 0; i < iterations; i++) {
            auto serialBegin = std::chrono::high_resolution_clock::now();
            diffuse(matrixSize, Axis::ZERO, value, oldValue, diff, dt);
            auto serialEnd = std::chrono::high_resolution_clock::now();
            auto serialTime = std::chrono::duration_cast<std::chrono::microseconds>(serialEnd - serialBegin).count();

            serialTimeMean += serialTime;


        }

        serialTimeMean /= iterations;
        std::cout << BOLD YELLOW "Diffuse: " << serialTimeMean << RESET " millis "<<std::endl<<std::endl;

        file << matrixSize << "," << serialTimeMean << ",";


        // OMP ----------------------------------------------------------------------------------------------
        int num_threads = omp_get_max_threads();
        omp_set_num_threads(num_threads);
        int threadMean = 0;
        int realThreadNumber = 0;

        // Calculate how many cells as maximum per thread
        const int max_rows = (int)(ceil((matrixSize-2) / num_threads) + 2);
        max_cells = max_rows * (matrixSize-2);        

        int64_t ompTimeMean = 0;

        for (int i = 0; i < iterations; i++) {    
            auto ompBegin = std::chrono::high_resolution_clock::now();
            omp_diffuse(matrixSize, Axis::ZERO, valueOmp, oldValueOmp, diff, dt, &realThreadNumber);
            auto ompEnd = std::chrono::high_resolution_clock::now();
            auto ompTime = std::chrono::duration_cast<std::chrono::microseconds>(ompEnd - ompBegin).count();
            threadMean += realThreadNumber;
            ompTimeMean += ompTime;
        }
        threadMean /= iterations;
        ompTimeMean /= iterations;
        
        std::cout << BOLD RED "OMP Diffuse: " << ompTimeMean << RESET " micros" << std::endl;

        speedup = (double) serialTimeMean / (double) ompTimeMean;
        efficiency = speedup / num_threads;
        std::cout << BOLD BLUE "Speedup: " << RESET << speedup << " ";
        std::cout << BOLD GREEN "Efficiency: " << RESET << efficiency << std::endl << std::endl;

        file << std::fixed << std::setprecision(2) << ompTimeMean << "," << std::setprecision(2) << speedup << "," << std::setprecision(2) << efficiency << "," << threadMean << "\n";
        
        printf("Speedup: %f\n", speedup);
    }
}

double randdouble() {
    return ((double) rand() / (RAND_MAX));
}

bool double_equals(double a, double b, double epsilon)
{
    return std::abs(a - b) < epsilon;
}

// Diffuse ----------------------------------------------------------------------------------------------

void diffuse(int N, Axis mode, std::vector<double> &value, std::vector<double> &oldValue, double diffusion, double dt) {
    double diffusionRate = dt * diffusion * N * N;

    lin_solve(N, mode, value, oldValue, diffusionRate);
}

void omp_diffuse(int N, Axis mode, std::vector<double> &value, std::vector<double> &oldValue, double diffusion, double dt, int * trdN) {
    double diffusionRate = dt * diffusion * N * N;

    omp_lin_solve(N, mode, value, oldValue, diffusionRate, trdN);
}

void cuda_diffuse(int N, Axis mode, std::vector<double> &value, std::vector<double> &oldValue, double diffusion, double dt) {
    double diffusionRate = dt * diffusion * N * N;
    
    dim3 BlockSize(16, 16, 1);
    dim3 GridSize((N+15)/16, (N+15)/16, 1);

    hipEvent_t cudaStart, cudaStop;	
    float milliseconds = 0;

    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaStop);

    double* d_value;
    double* d_oldValue;

    hipMalloc(&d_value, N * N * sizeof(double));
    hipMalloc(&d_oldValue, N * N * sizeof(double));

    hipMemcpy(d_value, &value[0], N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_oldValue, &oldValue[0], N * N * sizeof(double), hipMemcpyHostToDevice);

    std::cout<<"Hello from host"<<std::endl;

    hipEventRecord(cudaStart);    
    
    for (int i = 0; i < ITERATIONS; i++) 
        kernel_lin_solve<<<GridSize, BlockSize>>>(N, mode, &value[0], &oldValue[0], diffusionRate);

    hipEventRecord(cudaStop);

    hipMemcpy(&value[0], d_value, N * N * sizeof(double), hipMemcpyDeviceToHost);

    hipEventSynchronize(cudaStop);
    hipEventElapsedTime(&milliseconds, cudaStart, cudaStop);

    printf("Time for the kernel: %f ms\n", milliseconds);
    cudaTimeMean += milliseconds;


    hipFree(d_value);
    hipFree(d_oldValue);

    hipEventDestroy(cudaStart);
    hipEventDestroy(cudaStop);
}


// Advect ----------------------------------------------------------------------------------------------
void advect(int N, Axis mode, std::vector<double> &value, std::vector<double> &oldValue, std::vector<double> &vX, std::vector<double> &vY, double dt) {
    double i0, i1, j0, j1;

    double dt0 = dt * (N - 2);

	double s0, s1, t0, t1;
	double tmp1, tmp2, x, y;

	double Ndouble = N - 2;

	for(int i = 1; i < N - 1; i++) {
		for(int j = 1; j < N - 1; j++) {
            double v1 = vX[index(i, j, N)];
            double v2 = vY[index(i, j, N)];
            tmp1 = dt0 * v1;
            tmp2 = dt0 * v2;
            x = (double) i - tmp1;
            y = (double) j - tmp2;

            if(x < 0.5f) x = 0.5f;
            if(x > Ndouble + 0.5f) x = Ndouble + 0.5f;
            i0 = floor(x);
            i1 = i0 + 1.0f;
            if(y < 0.5f) y = 0.5f;
            if(y > Ndouble + 0.5f) y = Ndouble + 0.5f;
            j0 = floor(y);
            j1 = j0 + 1.0f;

            s1 = x - i0;
            s0 = 1.0f - s1;
            t1 = y - j0;
            t0 = 1.0f - t1;

            int i0i = i0;
            int i1i = i1;
            int j0i = j0;
            int j1i = j1;

			value[index(i, j, N)] =
				s0 * (t0 * oldValue[index(i0i, j0i, N)] + t1 * oldValue[index(i0i, j1i, N)]) +
				s1 * (t0 * oldValue[index(i1i, j0i, N)] + t1 * oldValue[index(i1i, j1i, N)]);
            }
    }
	set_bnd(N, mode, value);

}


void omp_advect(int N, Axis mode, std::vector<double> &value, std::vector<double> &oldValue, std::vector<double> &vX, std::vector<double> &vY, double dt, int * trdN) {
    double Ndouble = N - 2;
    double dt0 = dt * (N - 2);

    #pragma omp parallel
    {
        *trdN = omp_get_num_threads();

        double i0, i1, j0, j1;
        double s0, s1, t0, t1;
        double tmp1, tmp2, x, y;

        /*NOTE: #pragma omp parallel for default(shared) collapse(2)
        non funge perché non devono essere tutte shared
        schedule(static, 1) peggiora l'esecuzione*/
        #pragma omp for collapse(2)
        for(int i = 1; i < N - 1; i++) {
            for(int j = 1; j < N - 1; j++) {
                double v1 = vX[index(i, j, N)];
                double v2 = vY[index(i, j, N)];
                tmp1 = dt0 * v1;
                tmp2 = dt0 * v2;
                x = (double) i - tmp1;
                y = (double) j - tmp2;

                if(x < 0.5f) x = 0.5f;
                if(x > Ndouble + 0.5f) x = Ndouble + 0.5f;
                i0 = floor(x);
                i1 = i0 + 1.0f;
                if(y < 0.5f) y = 0.5f;
                if(y > Ndouble + 0.5f) y = Ndouble + 0.5f;
                j0 = floor(y);
                j1 = j0 + 1.0f;

                s1 = x - i0;
                s0 = 1.0f - s1;
                t1 = y - j0;
                t0 = 1.0f - t1;

                int i0i = i0;
                int i1i = i1;
                int j0i = j0;
                int j1i = j1;

                value[index(i, j, N)] =
                    s0 * (t0 * oldValue[index(i0i, j0i, N)] + t1 * oldValue[index(i0i, j1i, N)]) +
                    s1 * (t0 * oldValue[index(i1i, j0i, N)] + t1 * oldValue[index(i1i, j1i, N)]);
            }
        }
    
        omp_set_bnd(N, mode, value);
    }
}

// Project ----------------------------------------------------------------------------------------------

void project(int N, std::vector<double> &vX, std::vector<double> &vY, std::vector<double> &p, std::vector<double> &div) {
    for (uint32_t i = 1; i < N - 1; i++) {
        for (uint32_t j = 1; j < N - 1; j++) {
            div[index(i, j, N)] = -0.5f * (
                                vX[index(i + 1, j, N)]
                              - vX[index(i - 1, j, N)]
                              + vY[index(i, j + 1, N)]
                              - vY[index(i, j - 1, N)]
                        ) / N;
            p[index(i, j, N)] = 0;
        }
    }
    set_bnd(N, Axis::ZERO, div);
    set_bnd(N, Axis::ZERO, p);
    lin_solve(N, Axis::ZERO, p, div, 1);

    for (uint32_t i = 1; i < N - 1; i++) {
        for (uint32_t j = 1; j < N - 1; j++) {
            vX[index(i, j, N)] -= 0.5f * (p[index(i + 1, j, N)] - p[index(i - 1, j, N)]) * N;
            vY[index(i, j, N)] -= 0.5f * (p[index(i, j + 1, N)] - p[index(i, j - 1, N)]) * N;
        }
    }
    set_bnd(N, Axis::X, vX);
    set_bnd(N, Axis::Y, vY);
}


// Lin Solve ----------------------------------------------------------------------------------------------

void lin_solve(int N, Axis mode, std::vector<double> &nextValue, std::vector<double> &value, double diffusionRate) {
    double c = 1 + 4 * diffusionRate;
    double cRecip = 1.0 / c;
    for (int k = 0; k < ITERATIONS; k++) {
        for (int i = 1; i < N - 1; i++) {
            for (int j = 1; j < N - 1; j++) {
                nextValue[IX(i, j)] = (value[IX(i, j)]
                                   + diffusionRate * (
                        nextValue[IX(i + 1, j)]
                        + nextValue[IX(i - 1, j)]
                        + nextValue[IX(i, j + 1)]
                        + nextValue[IX(i, j - 1)]
                )) * cRecip;
            }
        }
        set_bnd(N, mode, nextValue);
    }
}

void omp_lin_solve(int N, Axis mode, std::vector<double> &nextValue, std::vector<double> &value, double diffusionRate, int * trdN) {
    double c = 1 + 4 * diffusionRate;
    double cRecip = 1.0 / c;
    for (int k = 0; k < ITERATIONS; k++)
    {
        
        #pragma omp parallel default(shared)
        {
            *trdN = omp_get_num_threads();
            #pragma omp for schedule(guided) collapse(2) 
            for (int i = 1; i < N - 1; i++)
            {
                for (int j = 1; j < N - 1; j++)
                {
                    
                    nextValue[IX(i, j)] = (value[IX(i, j)]
                                    + diffusionRate * (
                            nextValue[IX(i + 1, j)]
                            + nextValue[IX(i - 1, j)]
                            + nextValue[IX(i, j + 1)]
                            + nextValue[IX(i, j - 1)]
                    )) * cRecip;
                }
            }
            omp_set_bnd(N, mode, nextValue);
        }
        
    }
}

__global__ void kernel_lin_solve(int N, Axis mode, double* nextValue, double* value, double diffusionRate) {


    double c = 1 + 4 * diffusionRate;
    double cRecip = 1.0 / c;
    
    int col = threadIdx.x+blockIdx.x*blockDim.x;
	int row = threadIdx.y+blockIdx.y*blockDim.y;

	if(col == 0 || col >= N - 1 || row == 0 || row >= N - 1) return;

    printf("Hello from col: %d row: %d\n", col, row);

    nextValue[IX(row, col)] = (value[IX(row,col)]
                        + diffusionRate * (
            nextValue[IX(row + 1, col)]
            + nextValue[IX(row - 1, col)]
            + nextValue[IX(row, col + 1)]
            + nextValue[IX(row, col - 1)]
    )) * cRecip;

        // __syncthreads();
        // if (col == 1 && row == 1)
        //     kernel_set_bnd(N, mode, nextValue);
        // __syncthreads();

}

// Set Bnd ----------------------------------------------------------------------------------------------

void set_bnd(int N, Axis mode, std::vector<double> &attr) {
    for (int i = 1; i < N - 1; i++) {
        attr[IX(i, 0    )] = mode == Axis::Y ? -attr[IX(i, 1)] : attr[IX(i, 1)];
        attr[IX(i, N - 1)] = mode == Axis::Y ? -attr[IX(i, N - 2)] : attr[IX(i, N - 2)];
    }
    for (int j = 1; j < N - 1; j++) {
        attr[IX(0, j    )] = mode == Axis::X ? -attr[IX(1, j)] : attr[IX(1, j)];
        attr[IX(N - 1, j)] = mode == Axis::X ? -attr[IX(N - 2, j)] : attr[IX(N - 2, j)];
    }


    attr[IX(0    , 0    )] = 0.5f * (attr[IX(1, 0)] + attr[IX(0, 1)]);
    attr[IX(0    , N - 1)] = 0.5f * (attr[IX(1, N - 1)] + attr[IX(0, N - 2)]);

    attr[IX(N - 1, 0    )] = 0.5f * (attr[IX(N - 2, 0)] + attr[IX(N - 1, 1)]);
    attr[IX(N - 1, N - 1)] = 0.5f * (attr[IX(N - 2, N - 1)] + attr[IX(N - 1, N - 2)]);
}

/*i thread in omp_set_bound sono quelli utilizzati dalla funzione omp che la chiama (cioè non crea i suoi threads)*/
void omp_set_bnd(int N, Axis mode, std::vector<double> &attr) {
    #pragma omp for
    for (int i = 1; i < N - 1; i++) {
        attr[IX(i, 0    )] = mode == Axis::Y ? -attr[IX(i, 1)] : attr[IX(i, 1)];
        attr[IX(i, N - 1)] = mode == Axis::Y ? -attr[IX(i, N - 2)] : attr[IX(i, N - 2)];
    }
    #pragma omp for
    for (int j = 1; j < N - 1; j++) {
        attr[IX(0, j    )] = mode == Axis::X ? -attr[IX(1, j)] : attr[IX(1, j)];
        attr[IX(N - 1, j)] = mode == Axis::X ? -attr[IX(N - 2, j)] : attr[IX(N - 2, j)];
    }

    #pragma omp single
    {
        attr[IX(0    , 0    )] = 0.5f * (attr[IX(1, 0)] + attr[IX(0, 1)]);
        attr[IX(0    , N - 1)] = 0.5f * (attr[IX(1, N - 1)] + attr[IX(0, N - 2)]);

        attr[IX(N - 1, 0    )] = 0.5f * (attr[IX(N - 2, 0)] + attr[IX(N - 1, 1)]);
        attr[IX(N - 1, N - 1)] = 0.5f * (attr[IX(N - 2, N - 1)] + attr[IX(N - 1, N - 2)]);
    }
}

__device__ void kernel_set_bnd(int N, Axis mode, double *attr) {
    for (int i = 1; i < N - 1; i++) {
        attr[IX(i, 0    )] = mode == Axis::Y ? -attr[IX(i, 1)] : attr[IX(i, 1)];
        attr[IX(i, N - 1)] = mode == Axis::Y ? -attr[IX(i, N - 2)] : attr[IX(i, N - 2)];
    }
    for (int j = 1; j < N - 1; j++) {
        attr[IX(0, j    )] = mode == Axis::X ? -attr[IX(1, j)] : attr[IX(1, j)];
        attr[IX(N - 1, j)] = mode == Axis::X ? -attr[IX(N - 2, j)] : attr[IX(N - 2, j)];
    }


    attr[IX(0    , 0    )] = 0.5f * (attr[IX(1, 0)] + attr[IX(0, 1)]);
    attr[IX(0    , N - 1)] = 0.5f * (attr[IX(1, N - 1)] + attr[IX(0, N - 2)]);

    attr[IX(N - 1, 0    )] = 0.5f * (attr[IX(N - 2, 0)] + attr[IX(N - 1, 1)]);
    attr[IX(N - 1, N - 1)] = 0.5f * (attr[IX(N - 2, N - 1)] + attr[IX(N - 1, N - 2)]);
}