#include "hip/hip_runtime.h"
#include "fluid_matrix.hpp"
#include <sstream>

#define gpuErrchk(ans) gpuAssert((ans), __FILE__, __LINE__);

inline void gpuAssert(const hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::ostringstream oss;
        oss << "CUDA error: " << hipGetErrorString(code) << " at " << file << ":" << line;
        const std::string err_msg = oss.str();

        std::cerr << err_msg << std::endl;
        hipDeviceReset();
        throw std::runtime_error(err_msg);
    }
}

void FluidMatrix::CUDA_init() {
    const size_t size_bytes = this->size * this->size * sizeof(double);
    gpuErrchk(hipMalloc(&d_density, size_bytes));
    gpuErrchk(hipMalloc(&d_density_prev, size_bytes));
    gpuErrchk(hipMalloc(&d_vX, size_bytes));
    gpuErrchk(hipMalloc(&d_vX_prev, size_bytes));
    gpuErrchk(hipMalloc(&d_vY, size_bytes));
    gpuErrchk(hipMalloc(&d_vY_prev, size_bytes));
    gpuErrchk(hipMalloc(&d_newValue, size_bytes));
}

void FluidMatrix::CUDA_destroy() const {
    gpuErrchk(hipFree(d_density));
    gpuErrchk(hipFree(d_density_prev));
    gpuErrchk(hipFree(d_vX));
    gpuErrchk(hipFree(d_vX_prev));
    gpuErrchk(hipFree(d_vY));
    gpuErrchk(hipFree(d_vY_prev));
    gpuErrchk(hipFree(d_newValue));
}

void FluidMatrix::copyToHost() {
    const size_t size_bytes = this->size * this->size * sizeof(double);
    gpuErrchk(hipMemcpyAsync(density.data(), d_density, size_bytes, hipMemcpyDeviceToHost));
    // gpuErrchk(hipMemcpy(density_prev.data(), d_density_prev, size_bytes, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpyAsync(vX.data(), d_vX, size_bytes, hipMemcpyDeviceToHost));
    // gpuErrchk(hipMemcpy(vX_prev.data(), d_vX_prev, size_bytes, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpyAsync(vY.data(), d_vY, size_bytes, hipMemcpyDeviceToHost));
    // gpuErrchk(hipMemcpy(vY_prev.data(), d_vY_prev, size_bytes, hipMemcpyDeviceToHost));
}

void FluidMatrix::copyToDevice() const {
    const size_t size_bytes = this->size * this->size * sizeof(double);
    gpuErrchk(hipMemcpy(d_density, density.data(), size_bytes, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_density_prev, density_prev.data(), size_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_vX, vX.data(), size_bytes, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_vX_prev, vX_prev.data(), size_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_vY, vY.data(), size_bytes, hipMemcpyHostToDevice));
    // gpuErrchk(hipMemcpy(d_vY_prev, vY_prev.data(), size_bytes, hipMemcpyHostToDevice));
}


__device__ int index(const int i, const int j, const int size) { return i * size + j; }

__global__ void addVelocity_kernel(double *d_vX, double *d_vY, const int x, const int y, const int size, const double amountX, const double amountY) {
    d_vX[index(y, x, size)] += amountY;
    d_vY[index(y, x, size)] += amountX;
}

void FluidMatrix::CUDA_addVelocity(const int x, const int y, const double amountX, const double amountY) const {
    if (x < 0 || x >= size || y < 0 || y >= size) return;
    addVelocity_kernel<<<1, 1>>>(d_vX, d_vY, x, y, size, amountX, amountY);
}

__global__ void addDensity_kernel(double *d_density, const int x, const int y, const int size, const double amount) { d_density[index(y, x, size)] += amount; }

void FluidMatrix::CUDA_addDensity(const int x, const int y, const double amount) const {
    if (x < 0 || x >= size || y < 0 || y >= size) return;
    addDensity_kernel<<<1, 1>>>(d_density, x, y, size, amount);
}

void FluidMatrix::CUDA_reset() const {
    const size_t size_bytes = this->size * this->size * sizeof(double);
    gpuErrchk(hipMemset(d_density, 0, size_bytes));
    gpuErrchk(hipMemset(d_density_prev, 0, size_bytes));
    gpuErrchk(hipMemset(d_vX, 0, size_bytes));
    gpuErrchk(hipMemset(d_vX_prev, 0, size_bytes));
    gpuErrchk(hipMemset(d_vY, 0, size_bytes));
    gpuErrchk(hipMemset(d_vY_prev, 0, size_bytes));
}

__global__ void advect_kernel(const int size, double *d, const double *d0, const double *vX, const double *vY, const double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;
    int i0, j0, i1, j1;
    float x, y, s0, t0, s1, t1, dt0;

    dt0 = dt * (size - 2);

    x = i - dt0 * vX[index(i, j, size)];
    y = j - dt0 * vY[index(i, j, size)];
    if (x < 0.5f) x = 0.5f;
    if (x > size - 2 + 0.5f) x = size - 2 + 0.5f;
    i0 = (int) x;
    i1 = i0 + 1;
    if (y < 0.5f) y = 0.5f;
    if (y > size - 2 + 0.5f) y = size - 2 + 0.5f;
    j0 = (int) y;
    j1 = j0 + 1;
    s1 = x - i0;
    s0 = 1 - s1;
    t1 = y - j0;
    t0 = 1 - t1;
    d[index(i, j, size)] = s0 * (t0 * d0[index(i0, j0, size)] + t1 * d0[index(i0, j1, size)]) + s1 * (t0 * d0[index(i1, j0, size)] + t1 * d0[index(i1, j1, size)]);
}

__global__ void project_kernel(const int size, const double *vX, const double *vY, double *vY_prev) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    vY_prev[index(i, j, size)] = -0.5 * (vX[index(i + 1, j, size)] - vX[index(i - 1, j, size)] + vY[index(i, j + 1, size)] - vY[index(i, j - 1, size)]) * (size - 2);
}

__global__ void update_velocity_kernel(const int size, double *vX, double *vY, const double *vX_prev) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    vX[index(i, j, size)] -= 0.5 * (vX_prev[index(i + 1, j, size)] - vX_prev[index(i - 1, j, size)]) / (size - 2);
    vY[index(i, j, size)] -= 0.5 * (vX_prev[index(i, j + 1, size)] - vX_prev[index(i, j - 1, size)]) / (size - 2);
}

__global__ void set_bnd_edges(const int size, const Axis mode, double *attr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1 && i < size - 1) {
        attr[index(i, 0, size)] = mode == Y ? -attr[index(i, 1, size)] : attr[index(i, 1, size)];
        attr[index(i, size - 1, size)] = mode == Y ? -attr[index(i, size - 2, size)] : attr[index(i, size - 2, size)];
    }

    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j >= 1 && j < size - 1) {
        attr[index(0, j, size)] = mode == X ? -attr[index(1, j, size)] : attr[index(1, j, size)];
        attr[index(size - 1, j, size)] = mode == X ? -attr[index(size - 2, j, size)] : attr[index(size - 2, j, size)];
    }
}

__global__ void set_bnd_corners(const int size, double *attr) {
    attr[index(0, 0, size)] = 0.5f * (attr[index(1, 0, size)] + attr[index(0, 1, size)]);
    attr[index(0, size - 1, size)] = 0.5f * (attr[index(1, size - 1, size)] + attr[index(0, size - 2, size)]);
    attr[index(size - 1, 0, size)] = 0.5f * (attr[index(size - 2, 0, size)] + attr[index(size - 1, 1, size)]);
    attr[index(size - 1, size - 1, size)] = 0.5f * (attr[index(size - 2, size - 1, size)] + attr[index(size - 1, size - 2, size)]);
}

__global__ void lin_solve_kernel(const int size, const double *d_value, const double *d_oldValue, double *d_newValue, const double diffusionRate, const double cRecip) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    d_newValue[index(i, j, size)] = (d_oldValue[index(i, j, size)] + diffusionRate * (d_value[index(i + 1, j, size)] + d_value[index(i - 1, j, size)] +
                                                                                      d_value[index(i, j + 1, size)] + d_value[index(i, j - 1, size)])) *
                                    cRecip;
}

__global__ void fade_density_kernel(const int size, double *density) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 0 || i >= size * size) return;

    double d = density[i];
    density[i] = (d - 0.005f < 0) ? 0 : d - 0.005f;
}

void FluidMatrix::CUDA_step() {
    // Velocity
    {
        std::swap(d_vX_prev, d_vX);
        CUDA_diffuse(X, d_vX, d_vX_prev, visc, dt);

        std::swap(d_vY_prev, d_vY);
        CUDA_diffuse(Y, d_vY, d_vY_prev, visc, dt);

        CUDA_project(d_vX, d_vY, d_vX_prev, d_vY_prev);

        std::swap(d_vX_prev, d_vX);
        std::swap(d_vY_prev, d_vY);
        CUDA_advect(X, d_vX, d_vX_prev, d_vX_prev, d_vY_prev, dt);
        CUDA_advect(Y, d_vY, d_vY_prev, d_vX_prev, d_vY_prev, dt);

        CUDA_project(d_vX, d_vY, d_vX_prev, d_vY_prev);
    }

    // Density
    {
        std::swap(d_density_prev, d_density);
        CUDA_diffuse(ZERO, d_density, d_density_prev, visc, dt);

        std::swap(d_density_prev, d_density);
        CUDA_advect(ZERO, d_density, d_density_prev, d_vX, d_vY, dt);
    }

    CUDA_fadeDensity(d_density);

    CalculateVorticity(vX, vY, vorticity);
}

void FluidMatrix::CUDA_diffuse(const Axis mode, double *current, const double *previous, const double diffusion, const double dt) {
    double diffusionRate = dt * diffusion * (this->size - 2) * (this->size - 2);
    double cRecip = 1.0 / (1 + 4 * diffusionRate);
    CUDA_lin_solve(mode, current, previous, diffusionRate, cRecip);
}

void FluidMatrix::CUDA_advect(const Axis mode, double *d_density, const double *d_density0, const double *d_vX, const double *d_vY, const double dt) const {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    advect_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_density, d_density0, d_vX, d_vY, dt);
    gpuErrchk(hipPeekAtLastError());

    CUDA_set_bnd(mode, d_density);
}

void FluidMatrix::CUDA_project(double *d_vX, double *d_vY, double *d_vX_prev, double *d_vY_prev) {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    gpuErrchk(hipMemset(d_vX_prev, 0, size_bytes));
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    project_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX, d_vY, d_vY_prev);
    gpuErrchk(hipPeekAtLastError());

    CUDA_set_bnd(ZERO, d_vY_prev);
    CUDA_set_bnd(ZERO, d_vX_prev);
    // CUDA_lin_solve(ZERO, vX_prev, vY_prev, 1, 1.0 / 4);


    for (int k = 0; k < JACOBI_ITERATIONS; k++) {
        lin_solve_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX_prev, d_vY_prev, d_newValue, 1.0, 1.0 / 4);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        std::swap(d_vX_prev, d_newValue);
        CUDA_set_bnd(ZERO, d_vX_prev);
    }


    update_velocity_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX, d_vY, d_vX_prev);
    gpuErrchk(hipPeekAtLastError());

    CUDA_set_bnd(X, d_vX);
    CUDA_set_bnd(Y, d_vY);
}

__global__ void CUDA_set_bnd_kernel(const Axis mode, double *d_value, const int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= size || j >= size) return;

    // Top boundary (j == size-1) and bottom boundary (j == 0)
    if (j == 0 && i > 0 && i < size - 1) {
        // For vertical component (mode Y), reflect; otherwise copy
        d_value[index(i, 0, size)] = (mode == Axis::Y) ? -d_value[index(i, 1, size)] : d_value[index(i, 1, size)];
    }
    if (j == size - 1 && i > 0 && i < size - 1) {
        d_value[index(i, size - 1, size)] = (mode == Axis::Y) ? -d_value[index(i, size - 2, size)] : d_value[index(i, size - 2, size)];
    }

    // Left boundary (i == 0) and right boundary (i == size-1)
    if (i == 0 && j > 0 && j < size - 1) {
        d_value[index(0, j, size)] = (mode == Axis::X) ? -d_value[index(1, j, size)] : d_value[index(1, j, size)];
    }
    if (i == size - 1 && j > 0 && j < size - 1) {
        d_value[index(size - 1, j, size)] = (mode == Axis::X) ? -d_value[index(size - 2, j, size)] : d_value[index(size - 2, j, size)];
    }

    // Corners
    if (i == 0 && j == 0) {
        d_value[index(0, 0, size)] = 0.5 * (d_value[index(1, 0, size)] + d_value[index(0, 1, size)]);
    }
    if (i == 0 && j == size - 1) {
        d_value[index(0, size - 1, size)] = 0.5 * (d_value[index(1, size - 1, size)] + d_value[index(0, size - 2, size)]);
    }
    if (i == size - 1 && j == 0) {
        d_value[index(size - 1, 0, size)] = 0.5 * (d_value[index(size - 2, 0, size)] + d_value[index(size - 1, 1, size)]);
    }
    if (i == size - 1 && j == size - 1) {
        d_value[index(size - 1, size - 1, size)] = 0.5 * (d_value[index(size - 2, size - 1, size)] + d_value[index(size - 1, size - 2, size)]);
    }
}

void FluidMatrix::CUDA_set_bnd(const Axis mode, double *d_value) const {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x, (size + threadsPerBlock.y - 1) / threadsPerBlock.y);
    CUDA_set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(mode, d_value, this->size);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

void FluidMatrix::CUDA_lin_solve(const Axis mode, double *d_value, const double *d_oldValue, const double diffusionRate, const double cRecip) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    for (int k = 0; k < JACOBI_ITERATIONS; k++) {
        lin_solve_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_value, d_oldValue, d_newValue, diffusionRate, cRecip);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        std::swap(d_value, d_newValue);
        CUDA_set_bnd(mode, d_value);
    }
}

void FluidMatrix::CUDA_fadeDensity(double *d_density) const {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    fade_density_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_density);
    gpuErrchk(hipPeekAtLastError());
}
