#include "hip/hip_runtime.h"
#include "fluid_matrix.hpp"

#define gpuErrchk(ans)                                                                                                                                                             \
    {                                                                                                                                                                              \
        gpuAssert((ans), __FILE__, __LINE__);                                                                                                                                      \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
void FluidMatrix::CUDA_init() {
    const size_t size_bytes = this->size * this->size * sizeof(double);
    gpuErrchk(hipMalloc(&d_density, size_bytes));
    gpuErrchk(hipMalloc(&d_density_prev, size_bytes));
    gpuErrchk(hipMalloc(&d_vX, size_bytes));
    gpuErrchk(hipMalloc(&d_vX_prev, size_bytes));
    gpuErrchk(hipMalloc(&d_vY, size_bytes));
    gpuErrchk(hipMalloc(&d_vY_prev, size_bytes));
}

void FluidMatrix::CUDA_destroy() const {
    gpuErrchk(hipFree(d_density));
    gpuErrchk(hipFree(d_density_prev));
    gpuErrchk(hipFree(d_vX));
    gpuErrchk(hipFree(d_vX_prev));
    gpuErrchk(hipFree(d_vY));
    gpuErrchk(hipFree(d_vY_prev));
}

__device__ int index(const int i, const int j, const int size) { return i * size + j; }

__global__ void advect_kernel(int size, double *d, const double *d0, const double *vX, const double *vY, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    double dt0 = dt * (size - 2);
    double N_double = size - 2;

    double v1 = vX[index(i, j, size)];
    double v2 = vY[index(i, j, size)];

    double x = i - dt0 * v1;
    double y = j - dt0 * v2;

    if (x < 0.5) x = 0.5;
    if (x > N_double + 0.5) x = N_double + 0.5;
    if (y < 0.5) y = 0.5;
    if (y > N_double + 0.5) y = N_double + 0.5;

    int i0 = floor(x);
    int i1 = i0 + 1;
    int j0 = floor(y);
    int j1 = j0 + 1;

    double s1 = x - i0;
    double s0 = 1 - s1;
    double t1 = y - j0;
    double t0 = 1 - t1;

    d[index(i, j, size)] = s0 * (t0 * d0[index(i0, j0, size)] + t1 * d0[index(i0, j1, size)]) + s1 * (t0 * d0[index(i1, j0, size)] + t1 * d0[index(i1, j1, size)]);
}

__global__ void project_kernel(int size, double *vX, double *vY, double *div) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    div[index(i, j, size)] = -0.5 * (vX[index(i + 1, j, size)] - vX[index(i - 1, j, size)] + vY[index(i, j + 1, size)] - vY[index(i, j - 1, size)]) / size;
}

__global__ void update_velocity_kernel(int size, double *vX, double *vY, double *p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    vX[index(i, j, size)] -= 0.5 * (p[index(i + 1, j, size)] - p[index(i - 1, j, size)]) * size;
    vY[index(i, j, size)] -= 0.5 * (p[index(i, j + 1, size)] - p[index(i, j - 1, size)]) * size;
}

__global__ void set_bnd_edges(int size, Axis mode, double *attr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1 && i < size - 1) {
        attr[index(i, 0, size)] = mode == Y ? -attr[index(i, 1, size)] : attr[index(i, 1, size)];
        attr[index(i, size - 1, size)] = mode == Y ? -attr[index(i, size - 2, size)] : attr[index(i, size - 2, size)];
    }

    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j >= 1 && j < size - 1) {
        attr[index(0, j, size)] = mode == X ? -attr[index(1, j, size)] : attr[index(1, j, size)];
        attr[index(size - 1, j, size)] = mode == X ? -attr[index(size - 2, j, size)] : attr[index(size - 2, j, size)];
    }
}

__global__ void set_bnd_corners(int size, double *attr) {
    attr[index(0, 0, size)] = 0.5f * (attr[index(1, 0, size)] + attr[index(0, 1, size)]);
    attr[index(0, size - 1, size)] = 0.5f * (attr[index(1, size - 1, size)] + attr[index(0, size - 2, size)]);
    attr[index(size - 1, 0, size)] = 0.5f * (attr[index(size - 2, 0, size)] + attr[index(size - 1, 1, size)]);
    attr[index(size - 1, size - 1, size)] = 0.5f * (attr[index(size - 2, size - 1, size)] + attr[index(size - 1, size - 2, size)]);
}

__global__ void lin_solve_kernel(int size, double *value, const double *oldValue, double diffusionRate, double cRecip) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    value[index(i, j, size)] = (oldValue[index(i, j, size)] +
                                diffusionRate * (value[index(i - 1, j, size)] + value[index(i + 1, j, size)] + value[index(i, j - 1, size)] + value[index(i, j + 1, size)])) *
                               cRecip;
}

__global__ void fade_density_kernel(int size, double *density) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 0 || i >= size * size) return;

    double d = density[i];
    density[i] = (d - 0.005f < 0) ? 0 : d - 0.005f;
}

void FluidMatrix::CUDA_step() {
    // Velocity
    {
        SWAP(vX_prev, vX);
        CUDA_diffuse(X, vX, vX_prev, visc, dt);

        SWAP(vY_prev, vY);
        CUDA_diffuse(Y, vY, vY_prev, visc, dt);

        CUDA_project(vX, vY, vX_prev, vY_prev);

        SWAP(vX_prev, vX);
        SWAP(vY_prev, vY);
        CUDA_advect(X, vX, vX_prev, vX_prev, vY_prev, dt);
        CUDA_advect(Y, vY, vY_prev, vX_prev, vY_prev, dt);

        CUDA_project(vX, vY, vX_prev, vY_prev);
    }

    // Density
    {
        SWAP(density_prev, density);
        CUDA_diffuse(ZERO, density, density_prev, visc, dt);

        SWAP(density_prev, density);
        CUDA_advect(ZERO, density, density_prev, vX, vY, dt);
    }

    CUDA_fadeDensity(density);

    CalculateVorticity(vX, vY, vorticity);
}

void FluidMatrix::CUDA_diffuse(Axis mode, std::vector<double> &current, std::vector<double> &previous, double diffusion, double dt) const {
    double diffusionRate = dt * diffusion * (this->size - 2) * (this->size - 2);
    CUDA_lin_solve(mode, current, previous, diffusionRate);
}

void FluidMatrix::CUDA_advect(Axis mode, std::vector<double> &d, std::vector<double> &d0, std::vector<double> &vX, std::vector<double> &vY, double dt) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    double *dp_d, *dp_d0, *dp_vX, *dp_vY;

    switch (mode) {
        case X:
            dp_d = d_vX;
            dp_d0 = d_vX_prev;
            dp_vX = d_vX_prev;
            dp_vY = d_vY_prev;
            break;
        case Y:
            dp_d = d_vY;
            dp_d0 = d_vY_prev;
            dp_vX = d_vX_prev;
            dp_vY = d_vY_prev;
            break;
        case ZERO:
            dp_d = d_density;
            dp_d0 = d_density_prev;
            dp_vX = d_vX;
            dp_vY = d_vY;
            break;
        default: printf("Invalid mode\n"); return;
    }

    gpuErrchk(hipMemcpy(dp_d0, d0.data(), size_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dp_vX, vX.data(), size_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dp_vY, vY.data(), size_bytes, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    advect_kernel<<<numBlocks, threadsPerBlock>>>(this->size, dp_d, dp_d0, dp_vX, dp_vY, dt);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(d.data(), dp_d, size_bytes, hipMemcpyDeviceToHost));

    // CUDA_set_bnd(mode, d);
}

void FluidMatrix::CUDA_project(std::vector<double> &vX, std::vector<double> &vY, std::vector<double> &p, std::vector<double> &div) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    gpuErrchk(hipMemcpy(d_vX, vX.data(), size_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_vY, vY.data(), size_bytes, hipMemcpyHostToDevice));

    gpuErrchk(hipMemset(d_vX_prev, 0, size_bytes));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    project_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX, d_vY, d_vY_prev);
    gpuErrchk(hipPeekAtLastError());

    // CUDA_set_bnd(ZERO, div);
    // CUDA_set_bnd(ZERO, p);
    CUDA_lin_solve(ZERO, p, div, 1);

    update_velocity_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX, d_vY, d_vX_prev);
    gpuErrchk(hipPeekAtLastError());

    // CUDA_set_bnd(X, vX);
    // CUDA_set_bnd(Y, vY);

    gpuErrchk(hipMemcpy(vX.data(), d_vX, size_bytes, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(vY.data(), d_vY, size_bytes, hipMemcpyDeviceToHost));
}

void FluidMatrix::CUDA_set_bnd(Axis mode, std::vector<double> &attr) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    gpuErrchk(hipMemcpy(d_density, attr.data(), size_bytes, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    set_bnd_edges<<<numBlocks, threadsPerBlock>>>(this->size, mode, d_density);
    gpuErrchk(hipPeekAtLastError());
    set_bnd_corners<<<1, 1>>>(this->size, d_density);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(attr.data(), d_density, size_bytes, hipMemcpyDeviceToHost));
}

void FluidMatrix::CUDA_lin_solve(Axis mode, std::vector<double> &value, std::vector<double> &oldValue, double diffusionRate) const {

    const size_t size_bytes = this->size * this->size * sizeof(double);

    double c = diffusionRate;
    double cRecip = 1.0 / (1 + 4 * c);

    double *d_value, *d_oldValue;
    switch (mode) {
        case X:
            d_value = d_vX;
            d_oldValue = d_vX_prev;
            break;
        case Y:
            d_value = d_vY;
            d_oldValue = d_vY_prev;
            break;
        case ZERO:
            d_value = d_density;
            d_oldValue = d_density_prev;
            break;
        default: printf("Invalid mode\n"); return;
    }

    gpuErrchk(hipMemcpy(d_value, value.data(), size_bytes, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_oldValue, oldValue.data(), size_bytes, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    printf("CUDA diffusionRate: %f, cRecip: %f\n", diffusionRate, cRecip);
    printf("pre-CUDA Value[2][2]: %f\n", value[index(2, 2, this->size)]);
    printf("pre-CUDA Value[2][2-1]: %f\n", value[index(2, 1, this->size)]);
    printf("pre-CUDA Value[2][2+1]: %f\n", value[index(2, 3, this->size)]);
    printf("pre-CUDA Value[2-1][2]: %f\n", value[index(1, 2, this->size)]);
    printf("pre-CUDA Value[2+1][2]: %f\n", value[index(3, 2, this->size)]);

    for (int k = 0; k < ITERATIONS; k++) {
        lin_solve_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_value, d_oldValue, diffusionRate, cRecip);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        //CUDA_set_bnd(mode, value);
    }

    gpuErrchk(hipMemcpy(value.data(), d_value, size_bytes, hipMemcpyDeviceToHost));
    printf("post-CUDA Value[2][2]: %f\n", value[index(2, 2, this->size)]);
    printf("post-CUDA Value[2][2-1]: %f\n", value[index(2, 1, this->size)]);
    printf("post-CUDA Value[2][2+1]: %f\n", value[index(2, 3, this->size)]);
    printf("post-CUDA Value[2-1][2]: %f\n", value[index(1, 2, this->size)]);
    printf("post-CUDA Value[2+1][2]: %f\n", value[index(3, 2, this->size)]);
}

void FluidMatrix::CUDA_fadeDensity(std::vector<double> &density) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    gpuErrchk(hipMemcpy(d_density, density.data(), size_bytes, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    fade_density_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_density);
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(density.data(), d_density, size_bytes, hipMemcpyDeviceToHost));
}
