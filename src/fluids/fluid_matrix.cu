#include "hip/hip_runtime.h"
#include "fluid_matrix.hpp"

void FluidMatrix::CUDA_init() {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    hipMalloc(&d_density, size_bytes);
    hipMalloc(&d_density_prev, size_bytes);
    hipMalloc(&d_vX, size_bytes);
    hipMalloc(&d_vX_prev, size_bytes);
    hipMalloc(&d_vY, size_bytes);
    hipMalloc(&d_vY_prev, size_bytes);
    hipMalloc(&d_div, size_bytes);
    hipMalloc(&d_p, size_bytes);
}

void FluidMatrix::CUDA_destroy() const {
    hipFree(d_density);
    hipFree(d_density_prev);
    hipFree(d_vX);
    hipFree(d_vX_prev);
    hipFree(d_vY);
    hipFree(d_vY_prev);
    hipFree(d_div);
    hipFree(d_p);
}

__device__ int index(const int i, const int j, const int size) { return i * size + j; }

__global__ void advect_kernel(int size, double *d, const double *d0, const double *vX, const double *vY, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    double dt0 = dt * (size - 2);
    double N_double = size - 2;

    double v1 = vX[index(i, j, size)];
    double v2 = vY[index(i, j, size)];

    double x = i - dt0 * v1;
    double y = j - dt0 * v2;

    if (x < 0.5) x = 0.5;
    if (x > N_double + 0.5) x = N_double + 0.5;
    if (y < 0.5) y = 0.5;
    if (y > N_double + 0.5) y = N_double + 0.5;

    int i0 = floor(x);
    int i1 = i0 + 1;
    int j0 = floor(y);
    int j1 = j0 + 1;

    double s1 = x - i0;
    double s0 = 1 - s1;
    double t1 = y - j0;
    double t0 = 1 - t1;

    d[index(i, j, size)] = s0 * (t0 * d0[index(i0, j0, size)] + t1 * d0[index(i0, j1, size)]) + s1 * (t0 * d0[index(i1, j0, size)] + t1 * d0[index(i1, j1, size)]);
}

__global__ void project_kernel(int size, double *vX, double *vY, double *div) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    div[index(i, j, size)] = -0.5 * (vX[index(i + 1, j, size)] - vX[index(i - 1, j, size)] + vY[index(i, j + 1, size)] - vY[index(i, j - 1, size)]) / size;
}

__global__ void update_velocity_kernel(int size, double *vX, double *vY, double *p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    vX[index(i, j, size)] -= 0.5 * (p[index(i + 1, j, size)] - p[index(i - 1, j, size)]) * size;
    vY[index(i, j, size)] -= 0.5 * (p[index(i, j + 1, size)] - p[index(i, j - 1, size)]) * size;
}

__global__ void set_bnd_edges(int size, Axis mode, double *attr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 1 && i < size - 1) {
        attr[index(i, 0, size)] = mode == Y ? -attr[index(i, 1, size)] : attr[index(i, 1, size)];
        attr[index(i, size - 1, size)] = mode == Y ? -attr[index(i, size - 2, size)] : attr[index(i, size - 2, size)];
    }

    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j >= 1 && j < size - 1) {
        attr[index(0, j, size)] = mode == X ? -attr[index(1, j, size)] : attr[index(1, j, size)];
        attr[index(size - 1, j, size)] = mode == X ? -attr[index(size - 2, j, size)] : attr[index(size - 2, j, size)];
    }
}

__global__ void set_bnd_corners(int size, double *attr) {
    attr[index(0, 0, size)] = 0.5f * (attr[index(1, 0, size)] + attr[index(0, 1, size)]);
    attr[index(0, size - 1, size)] = 0.5f * (attr[index(1, size - 1, size)] + attr[index(0, size - 2, size)]);
    attr[index(size - 1, 0, size)] = 0.5f * (attr[index(size - 2, 0, size)] + attr[index(size - 1, 1, size)]);
    attr[index(size - 1, size - 1, size)] = 0.5f * (attr[index(size - 2, size - 1, size)] + attr[index(size - 1, size - 2, size)]);
}

__global__ void lin_solve_kernel(int size, double *value, const double *oldValue, double diffusionRate, double cRecip) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < 1 || i >= size - 1 || j < 1 || j >= size - 1) return;

    value[index(i, j, size)] = (oldValue[index(i, j, size)] +
                                diffusionRate * (value[index(i - 1, j, size)] + value[index(i + 1, j, size)] + value[index(i, j - 1, size)] + value[index(i, j + 1, size)])) *
                               cRecip;
}

__global__ void fade_density_kernel(int size, double *density) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 0 || i >= size * size) return;

    double d = density[i];
    density[i] = (d - 0.005f < 0) ? 0 : d - 0.005f;
}

void FluidMatrix::CUDA_step() {
    // Velocity
    {
        SWAP(vX_prev, vX);
        CUDA_diffuse(X, vX, vX_prev, visc, dt);

        SWAP(vY_prev, vY);
        CUDA_diffuse(Y, vY, vY_prev, visc, dt);

        CUDA_project(vX, vY, vX_prev, vY_prev);

        SWAP(vX_prev, vX);
        SWAP(vY_prev, vY);
        CUDA_advect(X, vX, vX_prev, vX_prev, vY_prev, dt);
        CUDA_advect(Y, vY, vY_prev, vX_prev, vY_prev, dt);

        CUDA_project(vX, vY, vX_prev, vY_prev);
    }

    // Density
    {
        SWAP(density_prev, density);
        CUDA_diffuse(ZERO, density, density_prev, diff, dt);

        SWAP(density_prev, density);
        CUDA_advect(ZERO, density, density_prev, vX, vY, dt);
    }

    CUDA_fadeDensity(density);

    CalculateVorticity(vX, vY, vorticity);
}

void FluidMatrix::CUDA_diffuse(Axis mode, std::vector<double> &current, std::vector<double> &previous, double diffusion, double dt) const {
    double diffusionRate = dt * diffusion * (this->size - 2) * (this->size - 2);
    CUDA_lin_solve(mode, current, previous, diffusionRate);
}

void FluidMatrix::CUDA_advect(Axis mode, std::vector<double> &d, std::vector<double> &d0, std::vector<double> &vX, std::vector<double> &vY, double dt) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    double *dp_d, *dp_d0, *dp_vX, *dp_vY;

    switch (mode) {
        case X:
            dp_d = d_vX;
            dp_d0 = d_vX_prev;
            dp_vX = d_vX_prev;
            dp_vY = d_vY_prev;
            break;
        case Y:
            dp_d = d_vY;
            dp_d0 = d_vY_prev;
            dp_vX = d_vX_prev;
            dp_vY = d_vY_prev;
            break;
        case ZERO:
            dp_d = d_density;
            dp_d0 = d_density_prev;
            dp_vX = d_vX_prev;
            dp_vY = d_vY_prev;
            break;
    }

    hipMemcpy(dp_d0, d0.data(), size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dp_vX, vX.data(), size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dp_vY, vY.data(), size_bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    advect_kernel<<<numBlocks, threadsPerBlock>>>(this->size, dp_d, dp_d0, dp_vX, dp_vY, dt);

    hipMemcpy(d.data(), dp_d, size_bytes, hipMemcpyDeviceToHost);

    CUDA_set_bnd(mode, d);
}

void FluidMatrix::CUDA_project(std::vector<double> &vX, std::vector<double> &vY, std::vector<double> &p, std::vector<double> &div) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    hipMemcpy(d_vX, vX.data(), size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vY, vY.data(), size_bytes, hipMemcpyHostToDevice);

    hipMemset(d_vX_prev, 0, size_bytes);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    project_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX, d_vY, d_vY_prev);

    CUDA_set_bnd(ZERO, div);
    CUDA_set_bnd(ZERO, p);
    CUDA_lin_solve(ZERO, p, div, 1);

    update_velocity_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_vX, d_vY, d_vX_prev);

    CUDA_set_bnd(X, vX);
    CUDA_set_bnd(Y, vY);

    hipMemcpy(vX.data(), d_vX, size_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(vY.data(), d_vY, size_bytes, hipMemcpyDeviceToHost);
}

void FluidMatrix::CUDA_set_bnd(Axis mode, std::vector<double> &attr) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    hipMemcpy(d_p, attr.data(), size_bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    set_bnd_edges<<<numBlocks, threadsPerBlock>>>(this->size, mode, d_p);
    set_bnd_corners<<<1, 1>>>(this->size, d_p);

    hipMemcpy(attr.data(), d_p, size_bytes, hipMemcpyDeviceToHost);
}

void FluidMatrix::CUDA_lin_solve(Axis mode, std::vector<double> &value, std::vector<double> &oldValue, double diffusionRate) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    double c = diffusionRate;
    double cRecip = 1.0 / (1 + 4 * c);

    double *d_value, *d_oldValue;
    switch (mode) {
        case X:
            d_value = d_vX;
            d_oldValue = d_vX_prev;
            break;
        case Y:
            d_value = d_vY;
            d_oldValue = d_vY_prev;
            break;
        case ZERO:
            d_value = d_density;
            d_oldValue = d_density_prev;
            c = 1;
            cRecip = 1.0 / 4;
            break;
    }

    hipMemcpy(d_value, value.data(), size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_oldValue, oldValue.data(), size_bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    for (int k = 0; k < ITERATIONS; k++) {
        lin_solve_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_value, d_oldValue, diffusionRate, cRecip);
        hipDeviceSynchronize();
        CUDA_set_bnd(mode, value);
    }

    hipMemcpy(value.data(), d_value, size_bytes, hipMemcpyDeviceToHost);
}

void FluidMatrix::CUDA_fadeDensity(std::vector<double> &density) const {
    const size_t size_bytes = this->size * this->size * sizeof(double);

    hipMemcpy(d_density, density.data(), size_bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((this->size + threadsPerBlock.x - 1) / threadsPerBlock.x, (this->size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    fade_density_kernel<<<numBlocks, threadsPerBlock>>>(this->size, d_density);

    hipMemcpy(density.data(), d_density, size_bytes, hipMemcpyDeviceToHost);
}
